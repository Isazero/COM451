#include "hip/hip_runtime.h"
/**************************************************************************
*
*     set up GPU for processing
*
**************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpu_main.h"

#include <hip/hip_runtime.h>

#define gScalar 0.2
texture<float, 2, hipReadModeElementType> texGreen;

/******************************************************************************/
GPU_Palette initGPUPalette(unsigned int imageWidth, unsigned int imageHeight)
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = ceil(imageWidth/32);  // however many blocks needed for image
  X.gBlocks.y = ceil(imageHeight/32);
  X.gBlocks.z = 1;

  X.palette_width = imageWidth;       // save this info
  X.palette_height = imageHeight;
  X.num_pixels = imageWidth * imageHeight;

  // allocate memory on GPU corresponding to pixel colors:
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.num_pixels * sizeof(float));
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  hipMalloc((void**) &X.green, X.num_pixels * sizeof(float)); // g
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  hipMalloc((void**) &X.blue, X.num_pixels * sizeof(float));  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  float *devPtr;
  size_t size=64*sizeof(float);
  hipMalloc((void **) &devPtr, size);
  
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  err = hipBindTexture(NULL, &texGreen, devPtr, &channelDesc, size);
  if (err != hipSuccess) {
    printf("cuda error bind texture = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return X;
}

/******************************************************************************/
void freeGPUPalette(GPU_Palette* P)
{
  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->blue);
}


/******************************************************************************/
int updatePalette(GPU_Palette* P, int xIdx, int yIdx, float z)
{

  updateReds <<< P->gBlocks, P->gThreads >>> (P->red, xIdx, yIdx, z);
  updateGreens <<< P->gBlocks, P->gThreads >>> (P->green, xIdx, yIdx, z);
	updateBlues <<< P->gBlocks, P->gThreads >>> (P->blue, xIdx, yIdx, z);

  return 0;
}

/******************************************************************************/
__global__ void updateReds(float* red, int xIdx, int yIdx, float z){

  int size = 5;
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  if ((xIdx < x + size) && (xIdx > x - size) && (yIdx < y + size) && (yIdx > y - size)) {
    red[vecIdx] = 1.0;
  } else {
    red[vecIdx] *= .98;
  }
}

/******************************************************************************/
__global__ void updateGreens(float* green, int xIdx, int yIdx, float z){
  int size = 5;
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);
  if ((xIdx < x + size) && (xIdx > x - size) && (yIdx < y + size) && (yIdx > y - size)) {
    green[vecIdx] = 0.5;
  } else {
    float acc = 0.0;
    for (int i = -5;i <= 5;i++) {
      for (int j = -5;j <= 5;j++) {
        acc += tex2D(texGreen, x + i, y + j);
      }
    }
    acc /= 121.0;
    green[vecIdx] = acc;
  }
}

/******************************************************************************/
__global__ void updateBlues(float* blue, int xIdx, int yIdx, float z){  
  int size = 5;
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);
  if ((xIdx < x + size) && (xIdx > x - size) && (yIdx < y + size) && (yIdx > y - size)) {
    blue[vecIdx] = 0.6;
  } else {
    blue[vecIdx] *= .93;
  }
}
